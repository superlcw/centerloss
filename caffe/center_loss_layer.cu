#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/center_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/gpu_util.cuh"

namespace caffe {
template <typename Dtype>
__global__ void CL_count_gpu(int nthreads, const int M, const Dtype* label, Dtype* count) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    count[index] = 0;
    for (int m = 0; m < M; m++) {
      const int label_value = static_cast<int>(label[m]);
      if (label_value == index) {
        count[index]++;
      }
    }
  }
}

template <typename Dtype>
__global__ void CL_difference_gpu(int nthreads, const int K, const Dtype* bottom,
        const Dtype* label, const Dtype* center, Dtype* difference) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int m = index / K;
    int k = index % K;
    const int label_value = static_cast<int>(label[m]);
    difference[index] = bottom[index] - center[label_value * K + k];
  }
}

template <typename Dtype>
__global__ void CL_L2_propagate_gpu(int nthreads, int K, const Dtype margin, const Dtype* difference,
        Dtype* propagate) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype d_norm = (Dtype)0.;
    for (int k = 0; k < K; k++) {
      d_norm += difference[index * K + k] * difference[index * K + k];
    }
    d_norm = sqrt(d_norm);
    propagate[index] = (d_norm - margin) / d_norm;
    propagate[index] = propagate[index] > (Dtype)0. ? propagate[index] : (Dtype)0.;
  }
}

template <typename Dtype>
__global__ void CL_cos_propagate_gpu(int nthreads, int K, const Dtype eps, const Dtype margin, const Dtype* bottom,
        const Dtype* label, const Dtype* center, Dtype* propagate) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // g, xx, xc, cc
    Dtype g = margin;
    Dtype r = (Dtype)0.;
    Dtype s = (Dtype)0.;
    Dtype t = (Dtype)0.;
    const int label_value = static_cast<int>(label[index]);
    for (int k = 0; k < K; k++) {
      r += bottom[index * K + k] * bottom[index * K + k];
      s += bottom[index * K + k] * center[label_value * K + k];
      t += center[label_value * K + k] * center[label_value * K + k];
    }
    if (!(s / sqrt(r) / sqrt(t) < g)) {
      propagate[index] = (Dtype)0.;
    } else {
      // a, b, c
      Dtype c = s * s - g * g * r * t;
      Dtype b = (Dtype)2. * (((Dtype)1. - g * g) * s * t - c);
      Dtype a = ((Dtype)1. - g * g) * t * (t - (Dtype)2. * s) + c;
      // x1, x2, propagate
      Dtype delta = b * b - (Dtype)4. * a * c;
      Dtype sqrt_delta = delta > (Dtype)0. ? sqrt(delta) : (Dtype)0.;
      Dtype x1 = (-b + sqrt_delta) / ((Dtype)2. * a);
      Dtype x2 = (-b - sqrt_delta) / ((Dtype)2. * a);
      if (x1*x2 < (Dtype)0. || x1+x2 > (Dtype)1.) {
        propagate[index] = (x1 > (Dtype)0. && x1 < (Dtype)1.000001) ? x1 : x2;
      } else {
        propagate[index] = x1 > x2 ? x1 : x2;
      }
    }
  }
}

template <typename Dtype>
__global__ void CL_propagated_difference_gpu(int nthreads, const int K, const Dtype* propagate,
        Dtype* difference) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int m = index / K;
    difference[index] = difference[index] * propagate[m];
  }
}

template <typename Dtype>
__global__ void CL_center_diff_gpu(int nthreads, const int K, const Dtype* label,
        const Dtype* count, const Dtype* difference, Dtype* center_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int m = index / K;
    int k = index % K;
    const int label_value = static_cast<int>(label[m]);
    caffe_gpu_atomic_add(-difference[index] / count[label_value], center_diff + label_value * K + k);
  }
}

template <typename Dtype>
void CenterLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int nthreads = N_;
  CL_count_gpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, M_, bottom[1]->gpu_data(), count_.mutable_gpu_data());
  // difference
  nthreads = M_ * K_;
  CL_difference_gpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, K_, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
                                this->blobs_[0]->gpu_data(), difference_.mutable_gpu_data());
  // propagate
  nthreads = M_;
  const Dtype eps_ = this->layer_param_.center_loss_param().eps();
  const Dtype margin_ = this->layer_param_.center_loss_param().margin();
  if (this->layer_param_.center_loss_param().type()==CenterLossParameter_DistanceType_L2) {
    CL_L2_propagate_gpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
       CAFFE_CUDA_NUM_THREADS>>>(nthreads, K_, margin_, difference_.gpu_data(),
                                 propagate_.mutable_gpu_data());
  } else if (this->layer_param_.center_loss_param().type()==CenterLossParameter_DistanceType_COSINE) {
    CL_cos_propagate_gpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
       CAFFE_CUDA_NUM_THREADS>>>(nthreads, K_, eps_, margin_, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
                                 this->blobs_[0]->gpu_data(), propagate_.mutable_gpu_data());
  }
  // propagated difference
  nthreads = M_ * K_;
  CL_propagated_difference_gpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
     CAFFE_CUDA_NUM_THREADS>>>(nthreads, K_, propagate_.gpu_data(), difference_.mutable_gpu_data());

  Dtype mean_propagate;
  caffe_gpu_asum(M_, propagate_.gpu_data(), &mean_propagate);
  Dtype dot;
  caffe_gpu_dot(M_ * K_, difference_.gpu_data(), difference_.gpu_data(), &dot);
  top[0]->mutable_cpu_data()[0] = dot / M_ / (Dtype)2.;
  top[1]->mutable_cpu_data()[0] = mean_propagate;
}

template <typename Dtype>
void CenterLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (this->param_propagate_down_[0]) {
    int nthreads = M_ * K_;
    CL_center_diff_gpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, K_, bottom[1]->gpu_data(), count_.gpu_data(),
                                  difference_.gpu_data(), this->blobs_[0]->mutable_gpu_diff());
  }

  if (propagate_down[0]) {
    caffe_gpu_scale(M_ * K_, top[0]->cpu_diff()[0] / M_, 
                             difference_.gpu_data(), bottom[0]->mutable_gpu_diff());
  }

  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CenterLossLayer);

}  // namespace caffe
